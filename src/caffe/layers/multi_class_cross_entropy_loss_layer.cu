#include "hip/hip_runtime.h"
#include "caffe/layers/multi_class_cross_entropy_loss_layer.hpp"
namespace caffe {
template <typename Dtype>
__global__ void MulticlassCrossEntropyLossForwardGPU(const int nthreads,
          const Dtype* input_data, const Dtype* label, Dtype* loss) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype tmp = input_data[index];
    if (int(label[index]) == 0) {
      Dtype tmp_inverse = 1 - tmp;
      if (tmp_inverse < 1e-10) {
        tmp_inverse = 1e-10;
      }
      loss[index] = -log(tmp_inverse);
      // loss[index] = -log(1 - tmp);
    }
    else {
      if (tmp < 1e-10) {
        tmp = 1e-10;
      }
      loss[index] = -log(tmp);
    }
  }
}
template <typename Dtype>
void MulticlassCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int nthreads = batch_size * channels;
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  // NOLINT_NEXT_LINE(whitespace/operators)
  MulticlassCrossEntropyLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, input_data, label, loss_data);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  loss /= batch_size;
  top[0]->mutable_cpu_data()[0] = loss;
}
template <typename Dtype>
__global__ void MulticlassCrossEntropyLossBackwardGPU(const int nthreads,
          const Dtype* input_data, const Dtype* label, Dtype* bottom_diff,
                  const int channels) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype tmp = input_data[index];
    if (int(label[index]) == 0) {
      Dtype tmp_inverse = 1 - tmp;
      if (tmp_inverse < 1e-10) {
        tmp_inverse = 1e-10;
      }
      bottom_diff[index] = 1.0 / (channels * (tmp_inverse));
    }
    else {
      if (tmp < 1e-10) {
        tmp = 1e-10;
      }
      bottom_diff[index] = -1.0 / (channels * tmp);
    }
  }
}
template <typename Dtype>
void MulticlassCrossEntropyLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* input_data = bottom[0]->gpu_data();
    const Dtype* label = bottom[1]->gpu_data();
    const int nthreads = batch_size * channels;

    MulticlassCrossEntropyLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, input_data, label, bottom_diff,
        channels);
    const Dtype loss_weight = top[0]->cpu_diff()[0];

    caffe_gpu_scal(bottom[0]->count(), loss_weight / batch_size, bottom_diff);
  }
}
INSTANTIATE_LAYER_GPU_FUNCS(MulticlassCrossEntropyLossLayer);
}  // namespace caffe
